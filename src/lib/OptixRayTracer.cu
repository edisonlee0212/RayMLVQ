#include "hip/hip_runtime.h"
#include <OptixRayTracer.hpp>
#include <optix_function_table_definition.h>
#include <FileUtil.hpp>

#include <glm/gtx/transform.hpp>
#define GL_TEXTURE_CUBE_MAP 0x8513
#include <cuda_gl_interop.h>
#include <RayDataDefinations.hpp>

using namespace RayMLVQ;

void OptixRayTracer::SetStatusChanged(const bool& value)
{
	m_statusChanged = value;
}

bool OptixRayTracer::RenderDebugOutput(const DefaultRenderingProperties& properties, std::vector<TriangleMesh>& meshes)
{
	if (properties.m_frameSize.x == 0 | properties.m_frameSize.y == 0) return true;
	if (!m_hasAccelerationStructure) return false;
	std::vector<std::pair<unsigned, hipTextureObject_t>> boundTextures;
	std::vector<hipGraphicsResource_t> boundResources;
	BuildShaderBindingTable(meshes, boundTextures, boundResources);
	if (m_debugRenderingPipeline.m_launchParams.m_defaultRenderingProperties.Changed(properties)) {
		m_debugRenderingPipeline.m_launchParams.m_defaultRenderingProperties = properties;
		m_statusChanged = true;
	}
	if (!m_accumulate || m_statusChanged) {
		m_debugRenderingPipeline.m_launchParams.m_frame.m_frameId = 0;
		m_statusChanged = false;
	}
#pragma region Bind texture
	hipArray_t outputArray;
	hipGraphicsResource_t outputTexture;
	hipArray_t environmentalMapPosXArray;
	hipArray_t environmentalMapNegXArray;
	hipArray_t environmentalMapPosYArray;
	hipArray_t environmentalMapNegYArray;
	hipArray_t environmentalMapPosZArray;
	hipArray_t environmentalMapNegZArray;
	hipGraphicsResource_t environmentalMapTexture;
#pragma region Bind output texture as cudaSurface
	CUDA_CHECK(GraphicsGLRegisterImage(&outputTexture, m_debugRenderingPipeline.m_launchParams.m_defaultRenderingProperties.m_outputTextureId, GL_TEXTURE_2D, hipGraphicsRegisterFlagsNone));
	CUDA_CHECK(GraphicsMapResources(1, &outputTexture, nullptr));
	CUDA_CHECK(GraphicsSubResourceGetMappedArray(&outputArray, outputTexture, 0, 0));
	// Specify surface
	struct hipResourceDesc hipResourceDesc;
	memset(&hipResourceDesc, 0, sizeof(hipResourceDesc));
	hipResourceDesc.resType = hipResourceTypeArray;
	// Create the surface objects
	hipResourceDesc.res.array.array = outputArray;
	// Create surface object
	CUDA_CHECK(CreateSurfaceObject(&m_debugRenderingPipeline.m_launchParams.m_frame.m_outputTexture, &hipResourceDesc));
#pragma endregion
#pragma region Bind environmental map as cudaTexture
	CUDA_CHECK(GraphicsGLRegisterImage(&environmentalMapTexture, m_debugRenderingPipeline.m_launchParams.m_defaultRenderingProperties.m_environmentalMapId, GL_TEXTURE_CUBE_MAP, hipGraphicsRegisterFlagsNone));
	CUDA_CHECK(GraphicsMapResources(1, &environmentalMapTexture, nullptr));
	CUDA_CHECK(GraphicsSubResourceGetMappedArray(&environmentalMapPosXArray, environmentalMapTexture, cudaGraphicsCubeFacePositiveX, 0));
	CUDA_CHECK(GraphicsSubResourceGetMappedArray(&environmentalMapNegXArray, environmentalMapTexture, cudaGraphicsCubeFaceNegativeX, 0));
	CUDA_CHECK(GraphicsSubResourceGetMappedArray(&environmentalMapPosYArray, environmentalMapTexture, cudaGraphicsCubeFacePositiveY, 0));
	CUDA_CHECK(GraphicsSubResourceGetMappedArray(&environmentalMapNegYArray, environmentalMapTexture, cudaGraphicsCubeFaceNegativeY, 0));
	CUDA_CHECK(GraphicsSubResourceGetMappedArray(&environmentalMapPosZArray, environmentalMapTexture, cudaGraphicsCubeFacePositiveZ, 0));
	CUDA_CHECK(GraphicsSubResourceGetMappedArray(&environmentalMapNegZArray, environmentalMapTexture, cudaGraphicsCubeFaceNegativeZ, 0));
	memset(&hipResourceDesc, 0, sizeof(hipResourceDesc));
	hipResourceDesc.resType = hipResourceTypeArray;
	struct hipTextureDesc hipTextureDesc;
	memset(&hipTextureDesc, 0, sizeof(hipTextureDesc));
	hipTextureDesc.addressMode[0] = hipAddressModeWrap;
	hipTextureDesc.addressMode[1] = hipAddressModeWrap;
	hipTextureDesc.filterMode = hipFilterModeLinear;
	hipTextureDesc.readMode = hipReadModeElementType;
	hipTextureDesc.normalizedCoords = 1;
	// Create texture object
	hipResourceDesc.res.array.array = environmentalMapPosXArray;
	CUDA_CHECK(CreateTextureObject(&m_debugRenderingPipeline.m_launchParams.m_skylight.m_environmentalMaps[0], &hipResourceDesc, &hipTextureDesc, nullptr));
	hipResourceDesc.res.array.array = environmentalMapNegXArray;
	CUDA_CHECK(CreateTextureObject(&m_debugRenderingPipeline.m_launchParams.m_skylight.m_environmentalMaps[1], &hipResourceDesc, &hipTextureDesc, nullptr));
	hipResourceDesc.res.array.array = environmentalMapPosYArray;
	CUDA_CHECK(CreateTextureObject(&m_debugRenderingPipeline.m_launchParams.m_skylight.m_environmentalMaps[2], &hipResourceDesc, &hipTextureDesc, nullptr));
	hipResourceDesc.res.array.array = environmentalMapNegYArray;
	CUDA_CHECK(CreateTextureObject(&m_debugRenderingPipeline.m_launchParams.m_skylight.m_environmentalMaps[3], &hipResourceDesc, &hipTextureDesc, nullptr));
	hipResourceDesc.res.array.array = environmentalMapPosZArray;
	CUDA_CHECK(CreateTextureObject(&m_debugRenderingPipeline.m_launchParams.m_skylight.m_environmentalMaps[4], &hipResourceDesc, &hipTextureDesc, nullptr));
	hipResourceDesc.res.array.array = environmentalMapNegZArray;
	CUDA_CHECK(CreateTextureObject(&m_debugRenderingPipeline.m_launchParams.m_skylight.m_environmentalMaps[5], &hipResourceDesc, &hipTextureDesc, nullptr));
#pragma endregion
#pragma endregion
#pragma region Upload parameters
	m_debugRenderingPipeline.m_launchParamsBuffer.Upload(&m_debugRenderingPipeline.m_launchParams, 1);
	m_debugRenderingPipeline.m_launchParams.m_frame.m_frameId++;
#pragma endregion
#pragma endregion
#pragma region Launch rays from camera
	OPTIX_CHECK(optixLaunch(/*! pipeline we're launching launch: */
		m_debugRenderingPipeline.m_pipeline, m_stream,
		/*! parameters and SBT */
		m_debugRenderingPipeline.m_launchParamsBuffer.DevicePointer(),
		m_debugRenderingPipeline.m_launchParamsBuffer.m_sizeInBytes,
		&m_debugRenderingPipeline.m_sbt,
		/*! dimensions of the launch: */
		m_debugRenderingPipeline.m_launchParams.m_defaultRenderingProperties.m_frameSize.x,
		m_debugRenderingPipeline.m_launchParams.m_defaultRenderingProperties.m_frameSize.y,
		1
	));
#pragma endregion
	CUDA_SYNC_CHECK();
#pragma region Remove texture binding.
	CUDA_CHECK(DestroySurfaceObject(m_debugRenderingPipeline.m_launchParams.m_frame.m_outputTexture));
	m_debugRenderingPipeline.m_launchParams.m_frame.m_outputTexture = 0;
	CUDA_CHECK(GraphicsUnmapResources(1, &outputTexture, 0));
	CUDA_CHECK(GraphicsUnregisterResource(outputTexture));

	CUDA_CHECK(DestroyTextureObject(m_debugRenderingPipeline.m_launchParams.m_skylight.m_environmentalMaps[0]));
	m_debugRenderingPipeline.m_launchParams.m_skylight.m_environmentalMaps[0] = 0;
	CUDA_CHECK(DestroyTextureObject(m_debugRenderingPipeline.m_launchParams.m_skylight.m_environmentalMaps[1]));
	m_debugRenderingPipeline.m_launchParams.m_skylight.m_environmentalMaps[1] = 0;
	CUDA_CHECK(DestroyTextureObject(m_debugRenderingPipeline.m_launchParams.m_skylight.m_environmentalMaps[2]));
	m_debugRenderingPipeline.m_launchParams.m_skylight.m_environmentalMaps[2] = 0;
	CUDA_CHECK(DestroyTextureObject(m_debugRenderingPipeline.m_launchParams.m_skylight.m_environmentalMaps[3]));
	m_debugRenderingPipeline.m_launchParams.m_skylight.m_environmentalMaps[3] = 0;
	CUDA_CHECK(DestroyTextureObject(m_debugRenderingPipeline.m_launchParams.m_skylight.m_environmentalMaps[4]));
	m_debugRenderingPipeline.m_launchParams.m_skylight.m_environmentalMaps[4] = 0;
	CUDA_CHECK(DestroyTextureObject(m_debugRenderingPipeline.m_launchParams.m_skylight.m_environmentalMaps[5]));
	m_debugRenderingPipeline.m_launchParams.m_skylight.m_environmentalMaps[5] = 0;

	CUDA_CHECK(GraphicsUnmapResources(1, &environmentalMapTexture, 0));
	CUDA_CHECK(GraphicsUnregisterResource(environmentalMapTexture));
#pragma endregion

	for (int i = 0; i < boundResources.size(); i++)
	{
		CUDA_CHECK(DestroySurfaceObject(boundTextures[i].second));
		CUDA_CHECK(GraphicsUnmapResources(1, &boundResources[i], 0));
		CUDA_CHECK(GraphicsUnregisterResource(boundResources[i]));
	}
	return true;
}

void OptixRayTracer::EstimateIllumination(const size_t& size, const IlluminationEstimationProperties& properties, CudaBuffer& lightProbes, std::vector<TriangleMesh>& meshes)
{
	if (!m_hasAccelerationStructure) return;
	std::vector<std::pair<unsigned, hipTextureObject_t>> boundTextures;
	std::vector<hipGraphicsResource_t> boundResources;
	BuildShaderBindingTable(meshes, boundTextures, boundResources);

#pragma region Upload parameters
	m_illuminationEstimationPipeline.m_launchParams.m_size = size;
	m_illuminationEstimationPipeline.m_launchParams.m_defaultIlluminationEstimationProperties = properties;
	m_illuminationEstimationPipeline.m_launchParams.m_lightProbes = reinterpret_cast<LightProbe<float>*>(lightProbes.DevicePointer());
	m_illuminationEstimationPipeline.m_launchParamsBuffer.Upload(&m_illuminationEstimationPipeline.m_launchParams, 1);
#pragma endregion
#pragma endregion
	if (size == 0)
	{
		std::cout << "Error!" << std::endl;
		return;
	}
#pragma region Launch rays from camera
	OPTIX_CHECK(optixLaunch(/*! pipeline we're launching launch: */
		m_illuminationEstimationPipeline.m_pipeline, m_stream,
		/*! parameters and SBT */
		m_illuminationEstimationPipeline.m_launchParamsBuffer.DevicePointer(),
		m_illuminationEstimationPipeline.m_launchParamsBuffer.m_sizeInBytes,
		&m_illuminationEstimationPipeline.m_sbt,
		/*! dimensions of the launch: */
		size,
		1,
		1
	));
#pragma endregion
	CUDA_SYNC_CHECK();
	for (int i = 0; i < boundResources.size(); i++)
	{
		CUDA_CHECK(DestroySurfaceObject(boundTextures[i].second));
		CUDA_CHECK(GraphicsUnmapResources(1, &boundResources[i], 0));
		CUDA_CHECK(GraphicsUnregisterResource(boundResources[i]));
	}
}

OptixRayTracer::OptixRayTracer()
{
	m_debugRenderingPipeline.m_launchParams.m_frame.m_frameId = 0;
	//std::cout << "#Optix: creating optix context ..." << std::endl;
	CreateContext();
	//std::cout << "#Optix: setting up module ..." << std::endl;
	CreateModule();
	//std::cout << "#Optix: creating raygen programs ..." << std::endl;
	CreateRayGenPrograms();
	//std::cout << "#Optix: creating miss programs ..." << std::endl;
	CreateMissPrograms();
	//std::cout << "#Optix: creating hitgroup programs ..." << std::endl;
	CreateHitGroupPrograms();
	//std::cout << "#Optix: setting up optix pipeline ..." << std::endl;
	AssemblePipelines();

	m_debugRenderingPipeline.m_launchParamsBuffer.Resize(sizeof(m_debugRenderingPipeline.m_launchParams));
	std::cout << "#Optix: context, module, pipeline, etc, all set up ..." << std::endl;
}

void OptixRayTracer::SetSkylightSize(const float& value)
{
	m_debugRenderingPipeline.m_launchParams.m_skylight.m_lightSize = value;
	m_statusChanged = true;
}

void OptixRayTracer::SetSkylightDir(const glm::vec3& value)
{
	m_debugRenderingPipeline.m_launchParams.m_skylight.m_direction = value;
	m_statusChanged = true;
}

static void context_log_cb(const unsigned int level,
	const char* tag,
	const char* message,
	void*)
{
	fprintf(stderr, "[%2d][%12s]: %s\n", static_cast<int>(level), tag, message);
}

void OptixRayTracer::CreateContext()
{
	// for this sample, do everything on one device
	const int deviceID = 0;
	CUDA_CHECK(StreamCreate(&m_stream));
	CUDA_CHECK(GetDeviceProperties(&m_deviceProps, deviceID));
	std::cout << "#Optix: running on device: " << m_deviceProps.name << std::endl;
	const hipError_t cuRes = hipCtxGetCurrent(&m_cudaContext);
	if (cuRes != hipSuccess)
		fprintf(stderr, "Error querying current context: error code %d\n", cuRes);
	OPTIX_CHECK(optixDeviceContextCreate(m_cudaContext, nullptr, &m_optixContext));
	OPTIX_CHECK(optixDeviceContextSetLogCallback
	(m_optixContext, context_log_cb, nullptr, 4));
}

extern "C" char DEFAULT_RENDERING_PTX[];
extern "C" char ILLUMINATION_ESTIMATION_PTX[];

extern "C" char RAYMLVQ_RENDERING_PTX[];

void OptixRayTracer::CreateModule()
{
	{
		m_debugRenderingPipeline.m_moduleCompileOptions.maxRegisterCount = 50;
		m_debugRenderingPipeline.m_moduleCompileOptions.optLevel = OPTIX_COMPILE_OPTIMIZATION_DEFAULT;
		m_debugRenderingPipeline.m_moduleCompileOptions.debugLevel = OPTIX_COMPILE_DEBUG_LEVEL_NONE;

		m_debugRenderingPipeline.m_pipelineCompileOptions = {};
		m_debugRenderingPipeline.m_pipelineCompileOptions.traversableGraphFlags = OPTIX_TRAVERSABLE_GRAPH_FLAG_ALLOW_SINGLE_GAS;
		m_debugRenderingPipeline.m_pipelineCompileOptions.usesMotionBlur = false;
		m_debugRenderingPipeline.m_pipelineCompileOptions.numPayloadValues = 2;
		m_debugRenderingPipeline.m_pipelineCompileOptions.numAttributeValues = 2;
		m_debugRenderingPipeline.m_pipelineCompileOptions.exceptionFlags = OPTIX_EXCEPTION_FLAG_NONE;
		m_debugRenderingPipeline.m_pipelineCompileOptions.pipelineLaunchParamsVariableName = "defaultRenderingLaunchParams";

		m_debugRenderingPipeline.m_pipelineLinkOptions.maxTraceDepth = 31;

		const std::string ptxCode = DEFAULT_RENDERING_PTX;

		char log[2048];
		size_t sizeof_log = sizeof(log);
		OPTIX_CHECK(optixModuleCreateFromPTX(m_optixContext,
			&m_debugRenderingPipeline.m_moduleCompileOptions,
			&m_debugRenderingPipeline.m_pipelineCompileOptions,
			ptxCode.c_str(),
			ptxCode.size(),
			log, &sizeof_log,
			&m_debugRenderingPipeline.m_module
		));
		if (sizeof_log > 1) std::cout << log << std::endl;
	}
	{
		m_illuminationEstimationPipeline.m_moduleCompileOptions.maxRegisterCount = 50;
		m_illuminationEstimationPipeline.m_moduleCompileOptions.optLevel = OPTIX_COMPILE_OPTIMIZATION_DEFAULT;
		m_illuminationEstimationPipeline.m_moduleCompileOptions.debugLevel = OPTIX_COMPILE_DEBUG_LEVEL_NONE;

		m_illuminationEstimationPipeline.m_pipelineCompileOptions = {};
		m_illuminationEstimationPipeline.m_pipelineCompileOptions.traversableGraphFlags = OPTIX_TRAVERSABLE_GRAPH_FLAG_ALLOW_SINGLE_GAS;
		m_illuminationEstimationPipeline.m_pipelineCompileOptions.usesMotionBlur = false;
		m_illuminationEstimationPipeline.m_pipelineCompileOptions.numPayloadValues = 2;
		m_illuminationEstimationPipeline.m_pipelineCompileOptions.numAttributeValues = 2;
		m_illuminationEstimationPipeline.m_pipelineCompileOptions.exceptionFlags = OPTIX_EXCEPTION_FLAG_NONE;
		m_illuminationEstimationPipeline.m_pipelineCompileOptions.pipelineLaunchParamsVariableName = "defaultIlluminationEstimationLaunchParams";

		m_illuminationEstimationPipeline.m_pipelineLinkOptions.maxTraceDepth = 31;

		const std::string ptxCode = ILLUMINATION_ESTIMATION_PTX;

		char log[2048];
		size_t sizeof_log = sizeof(log);
		OPTIX_CHECK(optixModuleCreateFromPTX(m_optixContext,
			&m_illuminationEstimationPipeline.m_moduleCompileOptions,
			&m_illuminationEstimationPipeline.m_pipelineCompileOptions,
			ptxCode.c_str(),
			ptxCode.size(),
			log, &sizeof_log,
			&m_illuminationEstimationPipeline.m_module
		));
		if (sizeof_log > 1) std::cout << log << std::endl;
	}
	{
		m_rayMLVQRenderingPipeline.m_moduleCompileOptions.maxRegisterCount = 50;
		m_rayMLVQRenderingPipeline.m_moduleCompileOptions.optLevel = OPTIX_COMPILE_OPTIMIZATION_DEFAULT;
		m_rayMLVQRenderingPipeline.m_moduleCompileOptions.debugLevel = OPTIX_COMPILE_DEBUG_LEVEL_NONE;

		m_rayMLVQRenderingPipeline.m_pipelineCompileOptions = {};
		m_rayMLVQRenderingPipeline.m_pipelineCompileOptions.traversableGraphFlags = OPTIX_TRAVERSABLE_GRAPH_FLAG_ALLOW_SINGLE_GAS;
		m_rayMLVQRenderingPipeline.m_pipelineCompileOptions.usesMotionBlur = false;
		m_rayMLVQRenderingPipeline.m_pipelineCompileOptions.numPayloadValues = 2;
		m_rayMLVQRenderingPipeline.m_pipelineCompileOptions.numAttributeValues = 2;
		m_rayMLVQRenderingPipeline.m_pipelineCompileOptions.exceptionFlags = OPTIX_EXCEPTION_FLAG_NONE;
		m_rayMLVQRenderingPipeline.m_pipelineCompileOptions.pipelineLaunchParamsVariableName = "rayMLVQRenderingLaunchParams";

		m_rayMLVQRenderingPipeline.m_pipelineLinkOptions.maxTraceDepth = 31;

		const std::string ptxCode = RAYMLVQ_RENDERING_PTX;

		char log[2048];
		size_t sizeof_log = sizeof(log);
		OPTIX_CHECK(optixModuleCreateFromPTX(m_optixContext,
			&m_rayMLVQRenderingPipeline.m_moduleCompileOptions,
			&m_rayMLVQRenderingPipeline.m_pipelineCompileOptions,
			ptxCode.c_str(),
			ptxCode.size(),
			log, &sizeof_log,
			&m_rayMLVQRenderingPipeline.m_module
		));
		if (sizeof_log > 1) std::cout << log << std::endl;
	}
}

void OptixRayTracer::CreateRayGenPrograms()
{
	{
		m_debugRenderingPipeline.m_rayGenProgramGroups.resize(1);
		OptixProgramGroupOptions pgOptions = {};
		OptixProgramGroupDesc pgDesc = {};
		pgDesc.kind = OPTIX_PROGRAM_GROUP_KIND_RAYGEN;
		pgDesc.raygen.module = m_debugRenderingPipeline.m_module;
		pgDesc.raygen.entryFunctionName = "__raygen__renderFrame";
		char log[2048];
		size_t sizeofLog = sizeof(log);
		OPTIX_CHECK(optixProgramGroupCreate(m_optixContext,
			&pgDesc,
			1,
			&pgOptions,
			log, &sizeofLog,
			&m_debugRenderingPipeline.m_rayGenProgramGroups[0]
		));
		if (sizeofLog > 1) std::cout << log << std::endl;
	}
	{
		m_illuminationEstimationPipeline.m_rayGenProgramGroups.resize(1);
		OptixProgramGroupOptions pgOptions = {};
		OptixProgramGroupDesc pgDesc = {};
		pgDesc.kind = OPTIX_PROGRAM_GROUP_KIND_RAYGEN;
		pgDesc.raygen.module = m_illuminationEstimationPipeline.m_module;
		pgDesc.raygen.entryFunctionName = "__raygen__illuminationEstimation";
		char log[2048];
		size_t sizeofLog = sizeof(log);
		OPTIX_CHECK(optixProgramGroupCreate(m_optixContext,
			&pgDesc,
			1,
			&pgOptions,
			log, &sizeofLog,
			&m_illuminationEstimationPipeline.m_rayGenProgramGroups[0]
		));
		if (sizeofLog > 1) std::cout << log << std::endl;
	}
	{
		m_rayMLVQRenderingPipeline.m_rayGenProgramGroups.resize(1);
		OptixProgramGroupOptions pgOptions = {};
		OptixProgramGroupDesc pgDesc = {};
		pgDesc.kind = OPTIX_PROGRAM_GROUP_KIND_RAYGEN;
		pgDesc.raygen.module = m_rayMLVQRenderingPipeline.m_module;
		pgDesc.raygen.entryFunctionName = "__raygen__renderFrame";
		char log[2048];
		size_t sizeofLog = sizeof(log);
		OPTIX_CHECK(optixProgramGroupCreate(m_optixContext,
			&pgDesc,
			1,
			&pgOptions,
			log, &sizeofLog,
			&m_rayMLVQRenderingPipeline.m_rayGenProgramGroups[0]
		));
		if (sizeofLog > 1) std::cout << log << std::endl;
	}
}

void OptixRayTracer::CreateMissPrograms()
{
	{
		m_debugRenderingPipeline.m_missProgramGroups.resize(static_cast<int>(DefaultRenderingRayType::RayTypeCount));
		char log[2048];
		size_t sizeofLog = sizeof(log);

		OptixProgramGroupOptions pgOptions = {};
		OptixProgramGroupDesc pgDesc = {};
		pgDesc.kind = OPTIX_PROGRAM_GROUP_KIND_MISS;
		pgDesc.miss.module = m_debugRenderingPipeline.m_module;

		// ------------------------------------------------------------------
		// radiance rays
		// ------------------------------------------------------------------
		pgDesc.miss.entryFunctionName = "__miss__radiance";

		OPTIX_CHECK(optixProgramGroupCreate(m_optixContext,
			&pgDesc,
			1,
			&pgOptions,
			log, &sizeofLog,
			&m_debugRenderingPipeline.m_missProgramGroups[static_cast<int>(DefaultRenderingRayType::RadianceRayType)]
		));
		if (sizeofLog > 1) std::cout << log << std::endl;
		// ------------------------------------------------------------------
		// shadow rays
		// ------------------------------------------------------------------
		pgDesc.miss.entryFunctionName = "__miss__shadow";
		OPTIX_CHECK(optixProgramGroupCreate(m_optixContext,
			&pgDesc,
			1,
			&pgOptions,
			log, &sizeofLog,
			&m_debugRenderingPipeline.m_missProgramGroups[static_cast<int>(DefaultRenderingRayType::ShadowRayType)]
		));
		if (sizeofLog > 1) std::cout << log << std::endl;
	}
	{
		m_illuminationEstimationPipeline.m_missProgramGroups.resize(static_cast<int>(DefaultIlluminationEstimationRayType::RayTypeCount));
		char log[2048];
		size_t sizeofLog = sizeof(log);

		OptixProgramGroupOptions pgOptions = {};
		OptixProgramGroupDesc pgDesc = {};
		pgDesc.kind = OPTIX_PROGRAM_GROUP_KIND_MISS;
		pgDesc.miss.module = m_illuminationEstimationPipeline.m_module;

		// ------------------------------------------------------------------
		// radiance rays
		// ------------------------------------------------------------------
		pgDesc.miss.entryFunctionName = "__miss__illuminationEstimation";

		OPTIX_CHECK(optixProgramGroupCreate(m_optixContext,
			&pgDesc,
			1,
			&pgOptions,
			log, &sizeofLog,
			&m_illuminationEstimationPipeline.m_missProgramGroups[static_cast<int>(DefaultIlluminationEstimationRayType::RadianceRayType)]
		));
		if (sizeofLog > 1) std::cout << log << std::endl;
	}
	{
		m_rayMLVQRenderingPipeline.m_missProgramGroups.resize(static_cast<int>(RayMLVQRenderingRayType::RayTypeCount));
		char log[2048];
		size_t sizeofLog = sizeof(log);

		OptixProgramGroupOptions pgOptions = {};
		OptixProgramGroupDesc pgDesc = {};
		pgDesc.kind = OPTIX_PROGRAM_GROUP_KIND_MISS;
		pgDesc.miss.module = m_rayMLVQRenderingPipeline.m_module;

		// ------------------------------------------------------------------
		// radiance rays
		// ------------------------------------------------------------------
		pgDesc.miss.entryFunctionName = "__miss__radiance";

		OPTIX_CHECK(optixProgramGroupCreate(m_optixContext,
			&pgDesc,
			1,
			&pgOptions,
			log, &sizeofLog,
			&m_rayMLVQRenderingPipeline.m_missProgramGroups[static_cast<int>(RayMLVQRenderingRayType::RadianceRayType)]
		));
		if (sizeofLog > 1) std::cout << log << std::endl;
	}
}

void OptixRayTracer::CreateHitGroupPrograms()
{
	{
		m_debugRenderingPipeline.m_hitGroupProgramGroups.resize(static_cast<int>(DefaultRenderingRayType::RayTypeCount));
		char log[2048];
		size_t sizeofLog = sizeof(log);

		OptixProgramGroupOptions pgOptions = {};
		OptixProgramGroupDesc pgDesc = {};
		pgDesc.kind = OPTIX_PROGRAM_GROUP_KIND_HITGROUP;
		pgDesc.hitgroup.moduleCH = m_debugRenderingPipeline.m_module;
		pgDesc.hitgroup.moduleAH = m_debugRenderingPipeline.m_module;
		// -------------------------------------------------------
		// radiance rays
		// -------------------------------------------------------
		pgDesc.hitgroup.entryFunctionNameCH = "__closesthit__radiance";
		pgDesc.hitgroup.entryFunctionNameAH = "__anyhit__radiance";
		OPTIX_CHECK(optixProgramGroupCreate(m_optixContext,
			&pgDesc,
			1,
			&pgOptions,
			log, &sizeofLog,
			&m_debugRenderingPipeline.m_hitGroupProgramGroups[static_cast<int>(DefaultRenderingRayType::RadianceRayType)]
		));
		if (sizeofLog > 1) std::cout << log << std::endl;

		// -------------------------------------------------------
		// shadow rays: technically we don't need this hit group,
		// since we just use the miss shader to check if we were not
		// in shadow
		// -------------------------------------------------------
		pgDesc.hitgroup.entryFunctionNameCH = "__closesthit__shadow";
		pgDesc.hitgroup.entryFunctionNameAH = "__anyhit__shadow";

		OPTIX_CHECK(optixProgramGroupCreate(m_optixContext,
			&pgDesc,
			1,
			&pgOptions,
			log, &sizeofLog,
			&m_debugRenderingPipeline.m_hitGroupProgramGroups[static_cast<int>(DefaultRenderingRayType::ShadowRayType)]
		));
		if (sizeofLog > 1) std::cout << log << std::endl;
	}
	{
		m_illuminationEstimationPipeline.m_hitGroupProgramGroups.resize(static_cast<int>(DefaultIlluminationEstimationRayType::RayTypeCount));
		char log[2048];
		size_t sizeofLog = sizeof(log);

		OptixProgramGroupOptions pgOptions = {};
		OptixProgramGroupDesc pgDesc = {};
		pgDesc.kind = OPTIX_PROGRAM_GROUP_KIND_HITGROUP;
		pgDesc.hitgroup.moduleCH = m_illuminationEstimationPipeline.m_module;
		pgDesc.hitgroup.moduleAH = m_illuminationEstimationPipeline.m_module;
		// -------------------------------------------------------
		// radiance rays
		// -------------------------------------------------------
		pgDesc.hitgroup.entryFunctionNameCH = "__closesthit__illuminationEstimation";
		pgDesc.hitgroup.entryFunctionNameAH = "__anyhit__illuminationEstimation";
		OPTIX_CHECK(optixProgramGroupCreate(m_optixContext,
			&pgDesc,
			1,
			&pgOptions,
			log, &sizeofLog,
			&m_illuminationEstimationPipeline.m_hitGroupProgramGroups[static_cast<int>(DefaultIlluminationEstimationRayType::RadianceRayType)]
		));
		if (sizeofLog > 1) std::cout << log << std::endl;
	}
	{
		m_rayMLVQRenderingPipeline.m_hitGroupProgramGroups.resize(static_cast<int>(RayMLVQRenderingRayType::RayTypeCount));
		char log[2048];
		size_t sizeofLog = sizeof(log);

		OptixProgramGroupOptions pgOptions = {};
		OptixProgramGroupDesc pgDesc = {};
		pgDesc.kind = OPTIX_PROGRAM_GROUP_KIND_HITGROUP;
		pgDesc.hitgroup.moduleCH = m_rayMLVQRenderingPipeline.m_module;
		pgDesc.hitgroup.moduleAH = m_rayMLVQRenderingPipeline.m_module;
		// -------------------------------------------------------
		// radiance rays
		// -------------------------------------------------------
		pgDesc.hitgroup.entryFunctionNameCH = "__closesthit__radiance";
		pgDesc.hitgroup.entryFunctionNameAH = "__anyhit__radiance";
		OPTIX_CHECK(optixProgramGroupCreate(m_optixContext,
			&pgDesc,
			1,
			&pgOptions,
			log, &sizeofLog,
			&m_rayMLVQRenderingPipeline.m_hitGroupProgramGroups[static_cast<int>(RayMLVQRenderingRayType::RadianceRayType)]
		));
		if (sizeofLog > 1) std::cout << log << std::endl;
	}
}

__global__ void ApplyTransformKernel(
	int size, glm::mat4 globalTransform,
	glm::vec3* positions, glm::vec3* normals, glm::vec3* tangents,
	glm::vec3* targetPositions, glm::vec3* targetNormals, glm::vec3* targetTangents)
{
	const int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < size)
	{
		targetPositions[idx] = globalTransform * glm::vec4(positions[idx], 1.0f);
		targetNormals[idx] = glm::normalize(globalTransform * glm::vec4(normals[idx], 0.0f));
		targetTangents[idx] = glm::normalize(globalTransform * glm::vec4(tangents[idx], 0.0f));
	}
}

void OptixRayTracer::BuildAccelerationStructure(std::vector<TriangleMesh>& meshes)
{
	bool uploadVertices = false;
	if (m_positionsBuffer.size() != meshes.size()) uploadVertices = true;
	else {
		for (auto& i : meshes)
		{
			if (i.m_verticesUpdateFlag) {
				uploadVertices = true;
				break;
			}
		}
	}
	if (uploadVertices) {
		for (auto& i : m_positionsBuffer) i.Free();
		for (auto& i : m_trianglesBuffer) i.Free();
		for (auto& i : m_normalsBuffer) i.Free();
		for (auto& i : m_tangentsBuffer) i.Free();
		for (auto& i : m_colorsBuffer) i.Free();
		for (auto& i : m_texCoordsBuffer) i.Free();
		for (auto& i : m_transformedPositionsBuffer) i.Free();
		for (auto& i : m_transformedNormalsBuffer) i.Free();
		for (auto& i : m_transformedTangentsBuffer) i.Free();

		m_positionsBuffer.clear();
		m_trianglesBuffer.clear();
		m_normalsBuffer.clear();
		m_tangentsBuffer.clear();
		m_colorsBuffer.clear();
		m_texCoordsBuffer.clear();
		m_transformedPositionsBuffer.clear();
		m_transformedNormalsBuffer.clear();
		m_transformedTangentsBuffer.clear();

		m_positionsBuffer.resize(meshes.size());
		m_trianglesBuffer.resize(meshes.size());
		m_normalsBuffer.resize(meshes.size());
		m_tangentsBuffer.resize(meshes.size());
		m_colorsBuffer.resize(meshes.size());
		m_texCoordsBuffer.resize(meshes.size());
		m_transformedTangentsBuffer.resize(meshes.size());
		m_transformedNormalsBuffer.resize(meshes.size());
		m_transformedPositionsBuffer.resize(meshes.size());
	}
	OptixTraversableHandle asHandle = 0;

	// ==================================================================
	// triangle inputs
	// ==================================================================
	std::vector<OptixBuildInput> triangleInput(meshes.size());
	std::vector<hipDeviceptr_t> deviceVertexPositions(meshes.size());
	std::vector<hipDeviceptr_t> deviceVertexTriangles(meshes.size());
	std::vector<hipDeviceptr_t> deviceTransforms(meshes.size());
	std::vector<uint32_t> triangleInputFlags(meshes.size());

	for (int meshID = 0; meshID < meshes.size(); meshID++) {
		// upload the model to the device: the builder
		TriangleMesh& triangleMesh = meshes[meshID];
		if (uploadVertices)
		{
			m_positionsBuffer[meshID].Upload(*triangleMesh.m_positions);
			m_tangentsBuffer[meshID].Upload(*triangleMesh.m_tangents);
			m_normalsBuffer[meshID].Upload(*triangleMesh.m_normals);
			m_transformedPositionsBuffer[meshID].Resize(triangleMesh.m_positions->size() * sizeof(glm::vec3));
			m_transformedNormalsBuffer[meshID].Resize(triangleMesh.m_normals->size() * sizeof(glm::vec3));
			m_transformedTangentsBuffer[meshID].Resize(triangleMesh.m_tangents->size() * sizeof(glm::vec3));
		}

		if (uploadVertices || triangleMesh.m_transformUpdateFlag) {
			int blockSize = 0;      // The launch configurator returned block size 
			int minGridSize = 0;    // The minimum grid size needed to achieve the maximum occupancy for a full device launch 
			int gridSize = 0;       // The actual grid size needed, based on input size
			int size = triangleMesh.m_positions->size();
			hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, ApplyTransformKernel, 0, size);
			gridSize = (size + blockSize - 1) / blockSize;
			ApplyTransformKernel << <gridSize, blockSize >> > (size, triangleMesh.m_globalTransform,
				static_cast<glm::vec3*>(m_positionsBuffer[meshID].m_dPtr), static_cast<glm::vec3*>(m_normalsBuffer[meshID].m_dPtr), static_cast<glm::vec3*>(m_tangentsBuffer[meshID].m_dPtr),
				static_cast<glm::vec3*>(m_transformedPositionsBuffer[meshID].m_dPtr), static_cast<glm::vec3*>(m_transformedNormalsBuffer[meshID].m_dPtr), static_cast<glm::vec3*>(m_transformedTangentsBuffer[meshID].m_dPtr));
			CUDA_SYNC_CHECK();
		}
		
		triangleMesh.m_verticesUpdateFlag = false;
		triangleMesh.m_transformUpdateFlag = false;

		m_texCoordsBuffer[meshID].Upload(*triangleMesh.m_texCoords);
		m_colorsBuffer[meshID].Upload(*triangleMesh.m_colors);
		m_trianglesBuffer[meshID].Upload(*triangleMesh.m_triangles);
		triangleInput[meshID] = {};
		triangleInput[meshID].type
			= OPTIX_BUILD_INPUT_TYPE_TRIANGLES;

		// create local variables, because we need a *pointer* to the
		// device pointers
		deviceVertexPositions[meshID] = m_transformedPositionsBuffer[meshID].DevicePointer();
		deviceVertexTriangles[meshID] = m_trianglesBuffer[meshID].DevicePointer();

		triangleInput[meshID].triangleArray.vertexFormat = OPTIX_VERTEX_FORMAT_FLOAT3;
		triangleInput[meshID].triangleArray.vertexStrideInBytes = sizeof(glm::vec3);
		triangleInput[meshID].triangleArray.numVertices = static_cast<int>(triangleMesh.m_positions->size());
		triangleInput[meshID].triangleArray.vertexBuffers = &deviceVertexPositions[meshID];

		//triangleInput[meshID].triangleArray.transformFormat = OPTIX_TRANSFORM_FORMAT_MATRIX_FLOAT12;
		//triangleInput[meshID].triangleArray.preTransform = deviceTransforms[meshID];

		triangleInput[meshID].triangleArray.indexFormat = OPTIX_INDICES_FORMAT_UNSIGNED_INT3;
		triangleInput[meshID].triangleArray.indexStrideInBytes = sizeof(glm::uvec3);
		triangleInput[meshID].triangleArray.numIndexTriplets = static_cast<int>(triangleMesh.m_triangles->size());
		triangleInput[meshID].triangleArray.indexBuffer = deviceVertexTriangles[meshID];

		triangleInputFlags[meshID] = 0;

		// in this example we have one SBT entry, and no per-primitive
		// materials:
		triangleInput[meshID].triangleArray.flags = &triangleInputFlags[meshID];
		triangleInput[meshID].triangleArray.numSbtRecords = 1;
		triangleInput[meshID].triangleArray.sbtIndexOffsetBuffer = 0;
		triangleInput[meshID].triangleArray.sbtIndexOffsetSizeInBytes = 0;
		triangleInput[meshID].triangleArray.sbtIndexOffsetStrideInBytes = 0;
	}
	// ==================================================================
	// BLAS setup
	// ==================================================================

	OptixAccelBuildOptions accelerateOptions = {};
	accelerateOptions.buildFlags = OPTIX_BUILD_FLAG_NONE
		| OPTIX_BUILD_FLAG_ALLOW_COMPACTION
		;
	accelerateOptions.motionOptions.numKeys = 1;
	accelerateOptions.operation = OPTIX_BUILD_OPERATION_BUILD;

	OptixAccelBufferSizes blasBufferSizes;
	OPTIX_CHECK(optixAccelComputeMemoryUsage
	(m_optixContext,
		&accelerateOptions,
		triangleInput.data(),
		static_cast<int>(meshes.size()),  // num_build_inputs
		&blasBufferSizes
	));

	// ==================================================================
	// prepare compaction
	// ==================================================================

	CudaBuffer compactedSizeBuffer;
	compactedSizeBuffer.Resize(sizeof(uint64_t));

	OptixAccelEmitDesc emitDesc;
	emitDesc.type = OPTIX_PROPERTY_TYPE_COMPACTED_SIZE;
	emitDesc.result = compactedSizeBuffer.DevicePointer();

	// ==================================================================
	// execute build (main stage)
	// ==================================================================

	CudaBuffer tempBuffer;
	tempBuffer.Resize(blasBufferSizes.tempSizeInBytes);

	CudaBuffer outputBuffer;
	outputBuffer.Resize(blasBufferSizes.outputSizeInBytes);

	OPTIX_CHECK(optixAccelBuild(m_optixContext,
		/* stream */nullptr,
		&accelerateOptions,
		triangleInput.data(),
		static_cast<int>(meshes.size()),
		tempBuffer.DevicePointer(),
		tempBuffer.m_sizeInBytes,

		outputBuffer.DevicePointer(),
		outputBuffer.m_sizeInBytes,

		&asHandle,

		&emitDesc, 1
	));
	CUDA_SYNC_CHECK();

	// ==================================================================
	// perform compaction
	// ==================================================================
	uint64_t compactedSize;
	compactedSizeBuffer.Download(&compactedSize, 1);

	m_acceleratedStructuresBuffer.Resize(compactedSize);
	OPTIX_CHECK(optixAccelCompact(m_optixContext,
		/*stream:*/nullptr,
		asHandle,
		m_acceleratedStructuresBuffer.DevicePointer(),
		m_acceleratedStructuresBuffer.m_sizeInBytes,
		&asHandle));
	CUDA_SYNC_CHECK();

	// ==================================================================
	// aaaaaand .... clean up
	// ==================================================================
	outputBuffer.Free(); // << the Uncompacted, temporary output buffer
	tempBuffer.Free();
	compactedSizeBuffer.Free();

	m_debugRenderingPipeline.m_launchParams.m_traversable = asHandle;
	m_illuminationEstimationPipeline.m_launchParams.m_traversable = asHandle;
	m_hasAccelerationStructure = true;
}

void OptixRayTracer::SetAccumulate(const bool& value)
{
	m_accumulate = value;
	m_statusChanged = true;
}

void OptixRayTracer::AssemblePipelines()
{
	
	{
		std::vector<OptixProgramGroup> programGroups;
		for (auto* pg : m_debugRenderingPipeline.m_rayGenProgramGroups)
			programGroups.push_back(pg);
		for (auto* pg : m_debugRenderingPipeline.m_missProgramGroups)
			programGroups.push_back(pg);
		for (auto* pg : m_debugRenderingPipeline.m_hitGroupProgramGroups)
			programGroups.push_back(pg);

		char log[2048];
		size_t sizeofLog = sizeof(log);
		OPTIX_CHECK(optixPipelineCreate(m_optixContext,
			&m_debugRenderingPipeline.m_pipelineCompileOptions,
			&m_debugRenderingPipeline.m_pipelineLinkOptions,
			programGroups.data(),
			static_cast<int>(programGroups.size()),
			log, &sizeofLog,
			&m_debugRenderingPipeline.m_pipeline
		));
		if (sizeofLog > 1) std::cout << log << std::endl;

		OPTIX_CHECK(optixPipelineSetStackSize
		(/* [in] The pipeline to configure the stack size for */
			m_debugRenderingPipeline.m_pipeline,
			/* [in] The direct stack size requirement for direct
			   callables invoked from IS or AH. */
			2 * 1024,
			/* [in] The direct stack size requirement for direct
			   callables invoked from RG, MS, or CH.  */
			2 * 1024,
			/* [in] The continuation stack requirement. */
			2 * 1024,
			/* [in] The maximum depth of a traversable graph
			   passed to trace. */
			1));
		if (sizeofLog > 1) std::cout << log << std::endl;
	}
	{
		std::vector<OptixProgramGroup> programGroups;
		for (auto* pg : m_illuminationEstimationPipeline.m_rayGenProgramGroups)
			programGroups.push_back(pg);
		for (auto* pg : m_illuminationEstimationPipeline.m_missProgramGroups)
			programGroups.push_back(pg);
		for (auto* pg : m_illuminationEstimationPipeline.m_hitGroupProgramGroups)
			programGroups.push_back(pg);

		char log[2048];
		size_t sizeofLog = sizeof(log);
		OPTIX_CHECK(optixPipelineCreate(m_optixContext,
			&m_illuminationEstimationPipeline.m_pipelineCompileOptions,
			&m_illuminationEstimationPipeline.m_pipelineLinkOptions,
			programGroups.data(),
			static_cast<int>(programGroups.size()),
			log, &sizeofLog,
			&m_illuminationEstimationPipeline.m_pipeline
		));
		if (sizeofLog > 1) std::cout << log << std::endl;

		OPTIX_CHECK(optixPipelineSetStackSize
		(/* [in] The pipeline to configure the stack size for */
			m_illuminationEstimationPipeline.m_pipeline,
			/* [in] The direct stack size requirement for direct
			   callables invoked from IS or AH. */
			2 * 1024,
			/* [in] The direct stack size requirement for direct
			   callables invoked from RG, MS, or CH.  */
			2 * 1024,
			/* [in] The continuation stack requirement. */
			2 * 1024,
			/* [in] The maximum depth of a traversable graph
			   passed to trace. */
			1));
		if (sizeofLog > 1) std::cout << log << std::endl;
	}
	{
		std::vector<OptixProgramGroup> programGroups;
		for (auto* pg : m_rayMLVQRenderingPipeline.m_rayGenProgramGroups)
			programGroups.push_back(pg);
		for (auto* pg : m_rayMLVQRenderingPipeline.m_missProgramGroups)
			programGroups.push_back(pg);
		for (auto* pg : m_rayMLVQRenderingPipeline.m_hitGroupProgramGroups)
			programGroups.push_back(pg);

		char log[2048];
		size_t sizeofLog = sizeof(log);
		OPTIX_CHECK(optixPipelineCreate(m_optixContext,
			&m_rayMLVQRenderingPipeline.m_pipelineCompileOptions,
			&m_rayMLVQRenderingPipeline.m_pipelineLinkOptions,
			programGroups.data(),
			static_cast<int>(programGroups.size()),
			log, &sizeofLog,
			&m_rayMLVQRenderingPipeline.m_pipeline
		));
		if (sizeofLog > 1) std::cout << log << std::endl;

		OPTIX_CHECK(optixPipelineSetStackSize
		(/* [in] The pipeline to configure the stack size for */
			m_rayMLVQRenderingPipeline.m_pipeline,
			/* [in] The direct stack size requirement for direct
			   callables invoked from IS or AH. */
			2 * 1024,
			/* [in] The direct stack size requirement for direct
			   callables invoked from RG, MS, or CH.  */
			2 * 1024,
			/* [in] The continuation stack requirement. */
			2 * 1024,
			/* [in] The maximum depth of a traversable graph
			   passed to trace. */
			1));
		if (sizeofLog > 1) std::cout << log << std::endl;
	}
}

void OptixRayTracer::BuildShaderBindingTable(std::vector<TriangleMesh>& meshes, std::vector<std::pair<unsigned, hipTextureObject_t>>& boundTextures, std::vector<hipGraphicsResource_t>& boundResources)
{
	{
		// ------------------------------------------------------------------
		// build raygen records
		// ------------------------------------------------------------------
		std::vector<DefaultRenderingRayGenRecord> raygenRecords;
		for (int i = 0; i < m_debugRenderingPipeline.m_rayGenProgramGroups.size(); i++) {
			DefaultRenderingRayGenRecord rec;
			OPTIX_CHECK(optixSbtRecordPackHeader(m_debugRenderingPipeline.m_rayGenProgramGroups[i], &rec));
			rec.m_data = nullptr; /* for now ... */
			raygenRecords.push_back(rec);
		}
		m_debugRenderingPipeline.m_rayGenRecordsBuffer.Upload(raygenRecords);
		m_debugRenderingPipeline.m_sbt.raygenRecord = m_debugRenderingPipeline.m_rayGenRecordsBuffer.DevicePointer();

		// ------------------------------------------------------------------
		// build miss records
		// ------------------------------------------------------------------
		std::vector<DefaultRenderingRayMissRecord> missRecords;
		for (int i = 0; i < m_debugRenderingPipeline.m_missProgramGroups.size(); i++) {
			DefaultRenderingRayMissRecord rec;
			OPTIX_CHECK(optixSbtRecordPackHeader(m_debugRenderingPipeline.m_missProgramGroups[i], &rec));
			rec.m_data = nullptr; /* for now ... */
			missRecords.push_back(rec);
		}
		m_debugRenderingPipeline.m_missRecordsBuffer.Upload(missRecords);
		m_debugRenderingPipeline.m_sbt.missRecordBase = m_debugRenderingPipeline.m_missRecordsBuffer.DevicePointer();
		m_debugRenderingPipeline.m_sbt.missRecordStrideInBytes = sizeof(DefaultRenderingRayMissRecord);
		m_debugRenderingPipeline.m_sbt.missRecordCount = static_cast<int>(missRecords.size());

		// ------------------------------------------------------------------
		// build hit records
		// ------------------------------------------------------------------

		// we don't actually have any objects in this example, but let's
		// create a dummy one so the SBT doesn't have any null pointers
		// (which the sanity checks in compilation would complain about)
		const int numObjects = m_positionsBuffer.size();
		std::vector<DefaultRenderingRayHitRecord> hitGroupRecords;
		for (int i = 0; i < numObjects; i++) {
			for (int rayID = 0; rayID < static_cast<int>(DefaultRenderingRayType::RayTypeCount); rayID++) {
				DefaultRenderingRayHitRecord rec;
				OPTIX_CHECK(optixSbtRecordPackHeader(m_debugRenderingPipeline.m_hitGroupProgramGroups[rayID], &rec));
				rec.m_data.m_position = reinterpret_cast<glm::vec3*>(m_transformedPositionsBuffer[i].DevicePointer());
				rec.m_data.m_triangle = reinterpret_cast<glm::uvec3*>(m_trianglesBuffer[i].DevicePointer());
				rec.m_data.m_normal = reinterpret_cast<glm::vec3*>(m_transformedNormalsBuffer[i].DevicePointer());
				rec.m_data.m_tangent = reinterpret_cast<glm::vec3*>(m_transformedTangentsBuffer[i].DevicePointer());
				rec.m_data.m_color = reinterpret_cast<glm::vec4*>(m_colorsBuffer[i].DevicePointer());
				rec.m_data.m_texCoord = reinterpret_cast<glm::vec2*>(m_texCoordsBuffer[i].DevicePointer());

				rec.m_data.m_surfaceColor = meshes[i].m_surfaceColor;
				rec.m_data.m_roughness = meshes[i].m_roughness;
				rec.m_data.m_metallic = meshes[i].m_metallic;
				rec.m_data.m_albedoTexture = 0;
				rec.m_data.m_normalTexture = 0;
				rec.m_data.m_diffuseIntensity = meshes[i].m_diffuseIntensity;
				if (meshes[i].m_albedoTexture != 0)
				{
					bool duplicate = false;
					for (auto& boundTexture : boundTextures)
					{
						if (boundTexture.first == meshes[i].m_albedoTexture)
						{
							rec.m_data.m_albedoTexture = boundTexture.second;
							duplicate = true;
							break;
						}
					}
					if (!duplicate) {
#pragma region Bind output texture
						hipArray_t textureArray;
						hipGraphicsResource_t graphicsResource;
						CUDA_CHECK(GraphicsGLRegisterImage(&graphicsResource, meshes[i].m_albedoTexture, GL_TEXTURE_2D, hipGraphicsRegisterFlagsReadOnly));
						CUDA_CHECK(GraphicsMapResources(1, &graphicsResource, nullptr));
						CUDA_CHECK(GraphicsSubResourceGetMappedArray(&textureArray, graphicsResource, 0, 0));
						struct hipResourceDesc hipResourceDesc;
						memset(&hipResourceDesc, 0, sizeof(hipResourceDesc));
						hipResourceDesc.resType = hipResourceTypeArray;
						hipResourceDesc.res.array.array = textureArray;
						struct hipTextureDesc hipTextureDesc;
						memset(&hipTextureDesc, 0, sizeof(hipTextureDesc));
						hipTextureDesc.addressMode[0] = hipAddressModeWrap;
						hipTextureDesc.addressMode[1] = hipAddressModeWrap;
						hipTextureDesc.filterMode = hipFilterModeLinear;
						hipTextureDesc.readMode = hipReadModeElementType;
						hipTextureDesc.normalizedCoords = 1;
						CUDA_CHECK(CreateTextureObject(&rec.m_data.m_albedoTexture, &hipResourceDesc, &hipTextureDesc, nullptr));
#pragma endregion
						boundResources.push_back(graphicsResource);
						boundTextures.emplace_back(meshes[i].m_albedoTexture, rec.m_data.m_albedoTexture);
					}
				}
				if (meshes[i].m_normalTexture != 0)
				{
					bool duplicate = false;
					for (auto& boundTexture : boundTextures)
					{
						if (boundTexture.first == meshes[i].m_normalTexture)
						{
							rec.m_data.m_normalTexture = boundTexture.second;
							duplicate = true;
							break;
						}
					}
					if (!duplicate) {
#pragma region Bind output texture
						hipArray_t textureArray;
						hipGraphicsResource_t graphicsResource;
						CUDA_CHECK(GraphicsGLRegisterImage(&graphicsResource, meshes[i].m_normalTexture, GL_TEXTURE_2D, hipGraphicsRegisterFlagsReadOnly));
						CUDA_CHECK(GraphicsMapResources(1, &graphicsResource, nullptr));
						CUDA_CHECK(GraphicsSubResourceGetMappedArray(&textureArray, graphicsResource, 0, 0));
						struct hipResourceDesc hipResourceDesc;
						memset(&hipResourceDesc, 0, sizeof(hipResourceDesc));
						hipResourceDesc.resType = hipResourceTypeArray;
						hipResourceDesc.res.array.array = textureArray;
						struct hipTextureDesc hipTextureDesc;
						memset(&hipTextureDesc, 0, sizeof(hipTextureDesc));
						hipTextureDesc.addressMode[0] = hipAddressModeWrap;
						hipTextureDesc.addressMode[1] = hipAddressModeWrap;
						hipTextureDesc.filterMode = hipFilterModeLinear;
						hipTextureDesc.readMode = hipReadModeElementType;
						hipTextureDesc.normalizedCoords = 1;
						CUDA_CHECK(CreateTextureObject(&rec.m_data.m_normalTexture, &hipResourceDesc, &hipTextureDesc, nullptr));
#pragma endregion
						boundResources.push_back(graphicsResource);
						boundTextures.emplace_back(meshes[i].m_normalTexture, rec.m_data.m_normalTexture);
					}
				}
				hitGroupRecords.push_back(rec);
			}
		}
		m_debugRenderingPipeline.m_hitGroupRecordsBuffer.Upload(hitGroupRecords);
		m_debugRenderingPipeline.m_sbt.hitgroupRecordBase = m_debugRenderingPipeline.m_hitGroupRecordsBuffer.DevicePointer();
		m_debugRenderingPipeline.m_sbt.hitgroupRecordStrideInBytes = sizeof(DefaultRenderingRayHitRecord);
		m_debugRenderingPipeline.m_sbt.hitgroupRecordCount = static_cast<int>(hitGroupRecords.size());
	}
	{
		// ------------------------------------------------------------------
		// build raygen records
		// ------------------------------------------------------------------
		std::vector<DefaultIlluminationEstimationRayGenRecord> raygenRecords;
		for (int i = 0; i < m_illuminationEstimationPipeline.m_rayGenProgramGroups.size(); i++) {
			DefaultIlluminationEstimationRayGenRecord rec;
			OPTIX_CHECK(optixSbtRecordPackHeader(m_illuminationEstimationPipeline.m_rayGenProgramGroups[i], &rec));
			rec.m_data = nullptr; /* for now ... */
			raygenRecords.push_back(rec);
		}
		m_illuminationEstimationPipeline.m_rayGenRecordsBuffer.Upload(raygenRecords);
		m_illuminationEstimationPipeline.m_sbt.raygenRecord = m_illuminationEstimationPipeline.m_rayGenRecordsBuffer.DevicePointer();

		// ------------------------------------------------------------------
		// build miss records
		// ------------------------------------------------------------------
		std::vector<DefaultIlluminationEstimationRayMissRecord> missRecords;
		for (int i = 0; i < m_illuminationEstimationPipeline.m_missProgramGroups.size(); i++) {
			DefaultIlluminationEstimationRayMissRecord rec;
			OPTIX_CHECK(optixSbtRecordPackHeader(m_illuminationEstimationPipeline.m_missProgramGroups[i], &rec));
			rec.m_data = nullptr; /* for now ... */
			missRecords.push_back(rec);
		}
		m_illuminationEstimationPipeline.m_missRecordsBuffer.Upload(missRecords);
		m_illuminationEstimationPipeline.m_sbt.missRecordBase = m_illuminationEstimationPipeline.m_missRecordsBuffer.DevicePointer();
		m_illuminationEstimationPipeline.m_sbt.missRecordStrideInBytes = sizeof(DefaultIlluminationEstimationRayMissRecord);
		m_illuminationEstimationPipeline.m_sbt.missRecordCount = static_cast<int>(missRecords.size());

		// ------------------------------------------------------------------
		// build hit records
		// ------------------------------------------------------------------

		// we don't actually have any objects in this example, but let's
		// create a dummy one so the SBT doesn't have any null pointers
		// (which the sanity checks in compilation would complain about)
		const int numObjects = m_positionsBuffer.size();
		std::vector<DefaultIlluminationEstimationRayHitRecord> hitGroupRecords;
		for (int i = 0; i < numObjects; i++) {
			for (int rayID = 0; rayID < static_cast<int>(DefaultIlluminationEstimationRayType::RayTypeCount); rayID++) {
				DefaultIlluminationEstimationRayHitRecord rec;
				OPTIX_CHECK(optixSbtRecordPackHeader(m_illuminationEstimationPipeline.m_hitGroupProgramGroups[rayID], &rec));
				rec.m_data.m_position = reinterpret_cast<glm::vec3*>(m_transformedPositionsBuffer[i].DevicePointer());
				rec.m_data.m_triangle = reinterpret_cast<glm::uvec3*>(m_trianglesBuffer[i].DevicePointer());
				rec.m_data.m_normal = reinterpret_cast<glm::vec3*>(m_transformedNormalsBuffer[i].DevicePointer());
				rec.m_data.m_tangent = reinterpret_cast<glm::vec3*>(m_transformedTangentsBuffer[i].DevicePointer());
				rec.m_data.m_color = reinterpret_cast<glm::vec4*>(m_colorsBuffer[i].DevicePointer());
				rec.m_data.m_texCoord = reinterpret_cast<glm::vec2*>(m_texCoordsBuffer[i].DevicePointer());

				rec.m_data.m_surfaceColor = meshes[i].m_surfaceColor;
				rec.m_data.m_roughness = meshes[i].m_roughness;
				rec.m_data.m_metallic = meshes[i].m_metallic;
				rec.m_data.m_albedoTexture = 0;
				rec.m_data.m_normalTexture = 0;
				rec.m_data.m_diffuseIntensity = meshes[i].m_diffuseIntensity;
				hitGroupRecords.push_back(rec);
			}
		}
		m_illuminationEstimationPipeline.m_hitGroupRecordsBuffer.Upload(hitGroupRecords);
		m_illuminationEstimationPipeline.m_sbt.hitgroupRecordBase = m_illuminationEstimationPipeline.m_hitGroupRecordsBuffer.DevicePointer();
		m_illuminationEstimationPipeline.m_sbt.hitgroupRecordStrideInBytes = sizeof(DefaultIlluminationEstimationRayHitRecord);
		m_illuminationEstimationPipeline.m_sbt.hitgroupRecordCount = static_cast<int>(hitGroupRecords.size());
	}
}
